#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <unordered_map>
#include <unordered_set>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <stack>

using namespace std;

struct Node {
    long long id;
    double lat, lon;
};

struct Edge {
    long long target;
    double length;
};

__device__ double haversine(double lat1, double lon1, double lat2, double lon2) {
    const double R = 6371000.0;
    double dLat = (lat2 - lat1) * M_PI / 180.0;
    double dLon = (lon2 - lon1) * M_PI / 180.0;
    lat1 *= M_PI / 180.0;
    lat2 *= M_PI / 180.0;
    double a = sin(dLat/2)*sin(dLat/2) + cos(lat1)*cos(lat2)*sin(dLon/2)*sin(dLon/2);
    double c = 2 * atan2(sqrt(a), sqrt(1-a));
    return R * c;
}

__global__ void expand_kernel(
    Edge* edges,
    Node* nodes,
    int* edge_offsets,
    long long* open_current,
    int open_size_current,
    long long* open_next,
    int* open_size_next,
    double* gScore,
    long long* cameFrom,
    long long goal,
    bool* goal_found,
    int num_nodes
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= open_size_current || *goal_found) return;

    long long current = open_current[idx];

    int start = edge_offsets[current];
    int end = (current + 1 < num_nodes) ? edge_offsets[current + 1] : edge_offsets[num_nodes];

    for (int e = start; e < end; ++e) {
        long long neighbor = edges[e].target;
        double tentative_g = gScore[current] + edges[e].length;

        if (tentative_g < gScore[neighbor]) {
            gScore[neighbor] = tentative_g;
            cameFrom[neighbor] = current;

            int pos = atomicAdd(open_size_next, 1);
            open_next[pos] = neighbor;

            if (neighbor == goal) {
                *goal_found = true;
            }
        }
    }
}

int main() {
    unordered_map<long long, Node> nodes_map;
    unordered_map<long long, vector<Edge>> graph;

    ifstream nodeFile("nodes_large.csv");
    string line;
    getline(nodeFile, line); // skip header
    while (getline(nodeFile, line)) {
        stringstream ss(line);
        string id_str, lat_str, lon_str;
        getline(ss, id_str, ',');
        getline(ss, lat_str, ',');
        getline(ss, lon_str, ',');
        long long id = stoll(id_str);
        double lat = stod(lat_str);
        double lon = stod(lon_str);
        nodes_map[id] = {id, lat, lon};
    }

    ifstream edgeFile("edges_large.csv");
    getline(edgeFile, line); // skip header
    while (getline(edgeFile, line)) {
        stringstream ss(line);
        string u_str, v_str, len_str;
        getline(ss, u_str, ',');
        getline(ss, v_str, ',');
        getline(ss, len_str, ',');
        long long u = stoll(u_str);
        long long v = stoll(v_str);
        double len = stod(len_str);
        graph[u].push_back({v, len});
        graph[v].push_back({u, len});
    }

    // Compress nodes into arrays
    vector<Node> nodes_vec;
    unordered_map<long long, int> id_to_idx;
    int idx = 0;
    for (auto& [id, node] : nodes_map) {
        id_to_idx[id] = idx++;
        nodes_vec.push_back(node);
    }

    int num_nodes = nodes_vec.size();
    vector<Edge> edges_vec;
    vector<int> edge_offsets(num_nodes + 1, 0);

    idx = 0;
    for (auto& node : nodes_vec) {
        edge_offsets[idx] = edges_vec.size();
        for (auto& e : graph[node.id]) {
            edges_vec.push_back({id_to_idx[e.target], e.length});
        }
        idx++;
    }
    edge_offsets[num_nodes] = edges_vec.size();

    long long start_id = 195386940;
    long long goal_id = 1616221113;

    if (id_to_idx.find(start_id) == id_to_idx.end() || id_to_idx.find(goal_id) == id_to_idx.end()) {
        cerr << "Start or goal not found!" << endl;
        return 1;
    }

    int start = id_to_idx[start_id];
    int goal = id_to_idx[goal_id];

    Edge* d_edges;
    Node* d_nodes;
    int* d_edge_offsets;
    long long* d_open_current;
    long long* d_open_next;
    int* d_open_size_next;
    double* d_gScore;
    long long* d_cameFrom;
    bool* d_goal_found;

    hipMalloc(&d_edges, edges_vec.size() * sizeof(Edge));
    hipMalloc(&d_nodes, nodes_vec.size() * sizeof(Node));
    hipMalloc(&d_edge_offsets, edge_offsets.size() * sizeof(int));
    hipMalloc(&d_open_current, num_nodes * sizeof(long long));
    hipMalloc(&d_open_next, num_nodes * sizeof(long long));
    hipMalloc(&d_open_size_next, sizeof(int));
    hipMalloc(&d_gScore, num_nodes * sizeof(double));
    hipMalloc(&d_cameFrom, num_nodes * sizeof(long long));
    hipMalloc(&d_goal_found, sizeof(bool));

    hipMemcpy(d_edges, edges_vec.data(), edges_vec.size() * sizeof(Edge), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes, nodes_vec.data(), nodes_vec.size() * sizeof(Node), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_offsets, edge_offsets.data(), edge_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    double* h_gScore = new double[num_nodes];
    long long* h_cameFrom = new long long[num_nodes];
    fill(h_gScore, h_gScore + num_nodes, 1e9);
    fill(h_cameFrom, h_cameFrom + num_nodes, -1);
    h_gScore[start] = 0.0;

    hipMemcpy(d_gScore, h_gScore, num_nodes * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cameFrom, h_cameFrom, num_nodes * sizeof(long long), hipMemcpyHostToDevice);

    long long h_open_current[num_nodes];
    h_open_current[0] = start;
    int h_open_size_current = 1;
    hipMemcpy(d_open_current, h_open_current, num_nodes * sizeof(long long), hipMemcpyHostToDevice);

    bool h_goal_found = false;
    hipMemcpy(d_goal_found, &h_goal_found, sizeof(bool), hipMemcpyHostToDevice);

    int iterations = 0;

    // Timing
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event);

    while (!h_goal_found && h_open_size_current > 0) {
        int zero = 0;
        hipMemcpy(d_open_size_next, &zero, sizeof(int), hipMemcpyHostToDevice);

        expand_kernel<<<(h_open_size_current + 255) / 256, 256>>>(
            d_edges, d_nodes, d_edge_offsets,
            d_open_current, h_open_size_current,
            d_open_next, d_open_size_next,
            d_gScore, d_cameFrom, goal, d_goal_found, num_nodes
        );
        hipDeviceSynchronize();

        hipMemcpy(&h_goal_found, d_goal_found, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(&h_open_size_current, d_open_size_next, sizeof(int), hipMemcpyDeviceToHost);

        swap(d_open_current, d_open_next);
        iterations++;
    }

    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);

    hipMemcpy(h_cameFrom, d_cameFrom, num_nodes * sizeof(long long), hipMemcpyDeviceToHost);
    hipMemcpy(h_gScore, d_gScore, num_nodes * sizeof(double), hipMemcpyDeviceToHost);

    cout << "\n🎯 Goal reached after " << iterations << " kernel launches.\n";

    cout << "Path:\n";
    vector<int> path_indices;
    for (int at = goal; at != -1; at = h_cameFrom[at]) {
        path_indices.push_back(at);
    }
    reverse(path_indices.begin(), path_indices.end());
    for (auto idx : path_indices) {
        cout << nodes_vec[idx].id << " ";
    }
    cout << endl;


    cout << "\n📏 Path length: " << h_gScore[goal] << " meters\n";
    cout << "⏱️ CUDA A* Execution time: " << milliseconds / 1000.0 << " seconds\n";

    hipFree(d_edges);
    hipFree(d_nodes);
    hipFree(d_edge_offsets);
    hipFree(d_open_current);
    hipFree(d_open_next);
    hipFree(d_open_size_next);
    hipFree(d_gScore);
    hipFree(d_cameFrom);
    hipFree(d_goal_found);

    delete[] h_gScore;
    delete[] h_cameFrom;

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    return 0;
}
